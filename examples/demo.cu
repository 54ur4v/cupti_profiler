#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <string>
#include <thrust/device_vector.h>

#include <cupti_profiler.h>

template<typename T>
__global__ void kernel(T begin, int size) {
  const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if(thread_id < size)
    *(begin + thread_id) += 1;
}

template<typename T>
void call_kernel(T& arg) {
  kernel<<<1, 100>>>(thrust::raw_pointer_cast(&arg[0]), arg.size());
}

int main() {
  using namespace std;
  //using namespace thrust;

  constexpr int N = 100;
  thrust::device_vector<float> data(N, 0);

  vector<string> event_names {
                              "active_warps",
                              "gst_inst_32bit",
                              "active_cycles"
                             };
  vector<string> metric_names {
                               "flop_count_dp",
                               "flop_count_sp",
                               "inst_executed"
                               //"stall_memory_throttle"
                              };

  cupti_profiler::profiler profiler(event_names, metric_names);

  // Get #passes required to compute all metrics and events
  const int passes = profiler.get_passes();

  profiler.start();
  //int passes = 1;
  for(int i=0; i<100; ++i) {
    call_kernel(data);
  }
  profiler.stop();

  profiler.print_event_values(std::cout, true);
  profiler.print_metric_values(std::cout, true);

  thrust::host_vector<float> h_data(data);

  printf("\n");
  for(int i = 0; i < 10; ++i) {
    printf("%lf ", h_data[i]);
  }
  printf("\n");
  return 0;
}
